#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

struct absupdateOutput_functor
{
  __device__ void operator()(float* output, const float* input) const
  {
    *output = abs(*input);
  }
};

void THNN_CudaLogSpace_add(THCState *state, THCudaTensor *input, THCudaTensor *output)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  THCudaTensor_resizeAs(state, output, input);
  THC_pointwiseApply2(state, output, input, absupdateOutput_functor());
}
