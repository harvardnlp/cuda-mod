#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"


struct logadd_functor
{
    __device__ void operator()(float* output, const float* input1, const float* input2) const
    {
        if (*input1 < *input2) {
            *output = log(1 + exp(*input1 - *input2)) + *input2;
        } else {
            *output = log(1 + exp(*input2 - *input1)) + *input1;
        }
    }
};

TH_API void THNN_CudaLogSpace_add(
    THCState *state, THCudaTensor *output,
    THCudaTensor *input1, THCudaTensor *input2) {
    THC_pointwiseApply3(state, output, input1, input2, logadd_functor());
}


struct logadd_inplace_functor
{
    __device__ void operator()(float* input1, const float* input2) const
    {
        if (*input1 < *input2) {
            *input1 = log(1 + exp(*input1 - *input2)) + *input2;
        } else {
            *input1 = log(1 + exp(*input2 - *input1)) + *input1;
        }
    }
};


TH_API void THNN_CudaLogSpace_add_inplace(
    THCState *state, THCudaTensor *input1, THCudaTensor *input2) {
    THC_pointwiseApply2(state, input1, input2, logadd_inplace_functor());
}

struct pointwisemod_functor
{
    __device__ void operator()(float* output, const float* input) const
  {
      unsigned int* inta = reinterpret_cast<unsigned int*>(output);
      // Set the last bit to the sign value;
      *inta = *inta ^ ((-((*input) == 1.0) ^ (*inta)) & (1 << 0));
      *output = *(reinterpret_cast<float*>(inta));

  }
};



struct signedAdd_functor
{
    __device__ void operator()(float* output, const float* input1, const float* input2) const
  {
      // Get back the signs
      float t1_sign = (((*reinterpret_cast<const unsigned int*>(input1)) >> 0) & 1) ? 1.0 : -1.0;
      float t2_sign = (((*reinterpret_cast<const unsigned int*>(input2)) >> 0) & 1) ? 1.0 : -1.0;


      // Do the add.
      float mx = max(*input1, *input2);
      float mn = min(*input1, *input2);
      float mn_mx = mn - mx;
      *output = log1p(exp(mn_mx) * t1_sign * t2_sign) + mx;

      // Change sign bit of output.
      float sign = (*input1 > *input2) ? t1_sign : t2_sign;
      unsigned int* inta = reinterpret_cast<unsigned int*>(output);
      *inta = *inta ^ ((-((sign) == 1.0) ^ (*inta)) & (1 << 0));
  }
};

struct getsign_functor
{
    __device__ void operator()(float* output, float* input) const
  {
      *output = (((*reinterpret_cast<const unsigned int*>(input) >> 0) & 1)) ? 1.0 : -1.0;
      // Reset the last bit
      unsigned int* inta = reinterpret_cast<unsigned int*>(input);
      *inta &= ~(1 << 0);

      // This is the nan trick.
      if (*output != *output) {
          *output = -1e10;
      }
  }
};

TH_API void THNN_CudaModSign(
    THCState *state, THCudaTensor *output, THCudaTensor *output_sign)
    {
    THC_pointwiseApply2(state, output, output_sign, pointwisemod_functor());
}

TH_API void THNN_CudaGetSign(
    THCState *state, THCudaTensor *output, THCudaTensor *output_sign)
    {
    THC_pointwiseApply2(state, output_sign, output, getsign_functor());
}


TH_API void THNN_CudaSignedLogSpace_add(
    THCState *state, THCudaTensor *output, THCudaTensor *output_sign,
    THCudaTensor *input1, THCudaTensor *input2,
    THCudaTensor *tensor1_sign, THCudaTensor *tensor2_sign) {

    THC_pointwiseApply2(state, input1, tensor1_sign, pointwisemod_functor());
    THC_pointwiseApply2(state, input2, tensor2_sign, pointwisemod_functor());

    THC_pointwiseApply3(state, output, input1, input2, signedAdd_functor());

    THC_pointwiseApply2(state, output_sign, output, getsign_functor());
}

struct signedAdd_inplace_functor
{
    __device__ void operator()(float* input1, const float* input2, const float* t1t2_sign_prod) const
  {
      float mx = max(*input1, *input2);
      float mn = min(*input1, *input2);
      float mn_mx = mn - mx;
      *input1 = log1p(exp(mn_mx) * *t1t2_sign_prod) + mx;
      if (*input1 != *input1) {
          *input1 = -1e10;
      }
      
  }
};

struct prod_functor
{
    __device__ void operator()(float* output, const float* input1, const float* input2) const
  {
      *output = *input1 * *input2;      
  }
};

struct ge_functor
{
    __device__ void operator()(float* tensor1_sign, const float* ge, const float* tensor2_sign) const
  {
      if (*ge < 1) {
          *tensor1_sign = *tensor2_sign;
      }      
  }
};

struct fixnan_functor
{
    __device__ void operator()(float* output) const
    {
      if (*output != *output) {
          *output = -1 * HIP_INF;
      }
    }
};

TH_API void THNN_CudaSignedLogSpace_add_inplace(
    THCState *state, THCudaTensor *input1, THCudaTensor *input2,
    THCudaTensor *tensor1_sign, THCudaTensor *tensor2_sign,
    THCudaTensor *t1t2_sign_prod, THCudaTensor *ge) {
    THC_pointwiseApply3(state, t1t2_sign_prod, tensor1_sign, tensor2_sign, prod_functor());
    THC_pointwiseApply3(state, tensor1_sign, ge, tensor2_sign, ge_functor());
    THC_pointwiseApply3(state, input1, input2, t1t2_sign_prod, signedAdd_inplace_functor());
}




TH_API void THNN_CudaFixNaN(
    THCState *state, THCudaTensor *input) {
    THC_pointwiseApply1(state, input, fixnan_functor());
}



// void THNN_CudaLogSpace_abs(THCState *state, THCudaTensor *output,
//                            THCudaTensor *input)
// {
//   THC_pointwiseApply2(state, output, input, abs_functor());
// }
// struct addexpOutput_functor
// {
//     const float max_;

//     addexpOutput_functor(float max)
//     : max_(max)
//   {}


//   __device__ void operator()(float* output, const float* input) const
//   {
//     *output = exp(*input - max_);
//   }
// };

// struct addexpOutputSign_functor
// {
//     const float max_;

//     addexpOutputSign_functor(float max)
//     : max_(max)
//   {}


//     __device__ void operator()(float* output, const float* input, const float* input_sign) const
//   {
//       *output = exp(*input - max_) * (*input_sign);
//   }
// };


// struct logaddOutput_functor
// {
//     const float max_;

//     logaddOutput_functor(float max)
//     : max_(max)
//   {}


//   __device__ void operator()(float* output, const float* input) const
//   {
//       *output = log(fabs(*input)) + max_;
//       if (*output != *output)
//           *output = -1e10;
//   }
// };

// void THNN_CudaLogSpace_bmm(THCState *state, THCudaTensor *output,
//                            THCudaTensor *input1, THCudaTensor *input2,
//                            THCudaTensor *tmp1, THCudaTensor *tmp2)
// {
//   THCUNN_assertSameGPU(state, 2, input1, output);
//   THCUNN_assertSameGPU(state, 2, input2, output);
//   THCUNN_assertSameGPU(state, 2, tmp1, output);
//   THCUNN_assertSameGPU(state, 2, tmp2, output);
//   // THCudaTensor_resizeAs(state, output, input1);
//   //find maxes
//   float max1 = THCudaTensor_maxall(state, input1);
//   float max2 = THCudaTensor_maxall(state, input2);

//   THC_pointwiseApply2(state, tmp1, input1, addexpOutput_functor(max1));
//   THC_pointwiseApply2(state, tmp2, input2, addexpOutput_functor(max2));

//   // call bmm
//   THCudaTensor_baddbmm(state, output, 0.0, output, 1.0, tmp1, tmp2);
//   THC_pointwiseApply2(state, output, output, logaddOutput_functor(max1 + max2));
// }


// void THNN_CudaSignedLogSpace_bmm(THCState *state, THCudaTensor *output, THCudaTensor *output_sign,
//                                  THCudaTensor *input1, THCudaTensor *input2,
//                                  THCudaTensor *tensor1_sign, THCudaTensor *tensor2_sign,
//                                  THCudaTensor *tmp1, THCudaTensor *tmp2)
// {
//   float max1 = THCudaTensor_maxall(state, input1);
//   float max2 = THCudaTensor_maxall(state, input2);

//   THC_pointwiseApply3(state, tmp1, input1, tensor1_sign, addexpOutputSign_functor(max1));
//   THC_pointwiseApply3(state, tmp2, input2, tensor2_sign, addexpOutputSign_functor(max2));


//   // call bmm
//   THCudaTensor_baddbmm(state, output, 0.0, output, 1.0, tmp1, tmp2);
//   THCudaTensor_sign(state, output_sign, output);
//   THC_pointwiseApply2(state, output, output, logaddOutput_functor(max1 + max2));
// }




// struct signedAdd_functor
// {
//     __device__ void operator()(float* output, const float* input1, const float* input2) const
//   {
//       float mx = max(*input1, *input2);
//       float mn = min(*input1, *input2);
//       float t1_sign = (((*reinterpret_cast<const unsigned int*>(input1)) >> 0) & 1) ? 1.0 : -1.0;
//       float t2_sign = (((*reinterpret_cast<const unsigned int*>(input2)) >> 0) & 1) ? 1.0 : -1.0;
//       float mn_mx = mn - mx;
//       *output = log1p(exp(mn_mx) * t1_sign * t2_sign) + mx;
//       float sign = (*input1 > *input2) ? t1_sign : t2_sign;

//       // Change sign bit of output.
//       unsigned int* inta = reinterpret_cast<unsigned int*>(output);
//       *inta = *inta ^ ((-((sign) == 1.0) ^ (*inta)) & (1 << 0));
//       // *output = *(reinterpret_cast<float*>(inta));
//   }
// };

// struct getsign_functor
// {
//     __device__ void operator()(float* output, float* input) const
//   {
//       *output = (((*reinterpret_cast<const unsigned int*>(input) >> 0) & 1)) ? 1.0 : -1.0;
//       // Reset the last bit
//       unsigned int* inta = reinterpret_cast<unsigned int*>(input);
//       *inta &= ~(1 << 0);
//       if (*output != *output) {
//           *output = -1e10;
//       }
//   }
// };



// TH_API void THNN_CudaSignedLogSpace_add(
//     THCState *state, THCudaTensor *output, THCudaTensor *output_sign,
//     THCudaTensor *input1, THCudaTensor *input2,
//     THCudaTensor *tensor1_sign, THCudaTensor *tensor2_sign) {

//     THC_pointwiseApply2(state, input1, tensor1_sign, pointwisemod_functor());
//     THC_pointwiseApply2(state, input2, tensor2_sign, pointwisemod_functor());

//     THC_pointwiseApply3(state, output, input1, input2, signedAdd_functor());

//     THC_pointwiseApply2(state, output_sign, output, getsign_functor());
// }


// TH_API void THNN_CudaSignedLogSpace_addSimple(
//     THCState *state, THCudaTensor *output,
//     THCudaTensor *input1, THCudaTensor *input2) {
//     THC_pointwiseApply3(state, output, input1, input2, signedAdd_functor());
// }




// TH_API void THNN_CudaSignedLogSpace_sum(
//     THCState *state, THCudaTensor *input, int dim) {
//     THC_pointwiseApply3(state, output, input1, input2, signedAdd_functor());
// }


// struct sum_functor
// {
//     const float max_;

//     logaddOutput_functor(float max)
//     : max_(max)
//     {}


//     __device__ void operator()(float* output, float* input) const
//   {
//       float sign = (((*reinterpret_cast<const unsigned int*>(input)) >> 0) & 1) ? 1.0 : -1.0;
//       *output = exp(tensor - max_) * sign;
//   }
// };


// TH_API void THNN_CudaSignedLogSpace_sumNumber(
//     THCState *state, THCudaTensor *input) {
//     float max1 = THCudaTensor_maxall(state, input1);

//     THC_pointwiseApply3(state, output, input1, input2, signedAdd_functor());
//     // float sum =
// }
