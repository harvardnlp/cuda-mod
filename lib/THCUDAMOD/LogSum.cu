#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

struct MaxFloat
{
  __device__ __forceinline__ float operator()(float max, float v) const
  {
    return fmaxf(max, v);
  }
};

struct SumFloat
{
  __device__ __forceinline__ float operator()(float sum, float v) const
  {
    return sum + v;
  }
};

struct SumExpFloat
{
  __device__ __forceinline__ SumExpFloat(float v)
    : max_k(v)
  {}

  __device__ __forceinline__ float operator()(float sum, float v) const
  {
    return sum + expf(v - max_k);
  }

  const float max_k;
};

struct NoFinal
{
  __device__ __forceinline__ float operator()(float v) const
  {
    return v;
  }
};

struct LSMFinal
{
  __device__ __forceinline__ LSMFinal(float m)
    : max_k(m)
  {}

  __device__ __forceinline__ float operator()(float v) const
  {
    return max_k + logf(v);
  }

  const float max_k;
};

template <typename Reduction, typename Finalize>
__device__ __forceinline__ float
blockReduce(float* smem, float val,
            const Reduction& r,
            float defaultVal,
            const Finalize& f)
{
  // To avoid RaW races from chaining blockReduce calls together, we
  // need a sync here
  __syncthreads();

  smem[threadIdx.x] = val;

  __syncthreads();

  float warpVal = defaultVal;

  // First warp will perform per-warp reductions for the remaining warps
  if ((threadIdx.x / 32) == 0) // only threads in warp1 go into this (if)
  {
    int lane = threadIdx.x % 32; // from 0 to 31

    // if less than 1024 threads per block, then only activate the relevant lanes
    if (lane < blockDim.x / 32)
    {
#pragma unroll
      for (int i = 0; i < 32; ++i)
      {
        warpVal = r(warpVal, smem[lane * 32 + i]);
      }

      smem[lane] = warpVal;
    }
  }

  __syncthreads();

  // First thread will perform a reduction of the above per-warp reductions
  float blockVal = defaultVal;

  if (threadIdx.x == 0)
  {
    for (int i = 0; i < blockDim.x / 32; ++i)
    {
      blockVal = r(blockVal, smem[i]);
    }

    smem[0] = f(blockVal);
  }

  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

template <typename Reduction>
__device__ __forceinline__ float
blockReduce(float* smem, float val,
            const Reduction& r,
            float defaultVal)
{
  return blockReduce<Reduction, NoFinal>(smem, val, r, defaultVal, NoFinal());
}

template <typename Reduction, int ILP>
__device__ __forceinline__ float
ilpReduce(float* data,
          int size,
          const Reduction& r,
          float defaultVal)
{
  float threadVal = defaultVal;
  int offset = threadIdx.x;

  int last = size % (ILP * blockDim.x);

  // Body (unroll by ILP times)
  for (; offset < size - last; offset += blockDim.x * ILP)
  {
    float tmp[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      tmp[j] = data[offset + j * blockDim.x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      threadVal = r(threadVal, tmp[j]);
    }
  }

  // Epilogue
  for (; offset < size; offset += blockDim.x)
  {
    threadVal = r(threadVal, data[offset]);
  }

  return threadVal;
}

template <int ILP>
__global__ void
cunn_LogSum_kernel(float *output, float *input, int elements)
{
  extern __shared__ float buffer[];
  // forward pointers to batch[blockIdx.x]
  // each block handles a sample in the mini-batch
  input += blockIdx.x * elements;

  // find the max of the batch
  float threadMax =
    ilpReduce<MaxFloat, ILP>(input, elements, MaxFloat(), -FLT_MAX);
  // find the max over all batches
  float max_k =
    blockReduce<MaxFloat>(buffer, threadMax, MaxFloat(), -FLT_MAX);

  float threadExp =
    ilpReduce<SumExpFloat, ILP>(input, elements, SumExpFloat(max_k), 0.0f);
  float logsum_k =
    blockReduce<SumFloat, LSMFinal>(
      buffer, threadExp, SumFloat(), 0.0f, LSMFinal(max_k));

    // This is the nan trick.
  if (logsum_k != logsum_k) {
      logsum_k = -HIP_INF_F;
  }

  output[blockIdx.x] = logsum_k;
}

// Sum over each batch
TH_API void THNN_CudaLogSpace_sum(
    THCState *state, THCudaTensor *output,
    THCudaTensor *input) {

  THCUNN_assertSameGPU(state, 2, input, output);
  input = THCudaTensor_newContiguous(state, input);
  THCudaTensor_resizeAs(state, output, input);

  int batchSize = THCudaTensor_size(state, input, 0);
  int elementSize = THCudaTensor_size(state, input, 1);

  dim3 grid(batchSize);
  dim3 block(1024);

  cunn_LogSum_kernel<2>
    <<<grid, block, block.x * sizeof(float), THCState_getCurrentStream(state)>>>(
      THCudaTensor_data(state, output),
      THCudaTensor_data(state, input),
      elementSize
  );
  THCudaCheck(hipGetLastError());
  THCudaTensor_free(state, input);
}
